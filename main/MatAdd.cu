#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

__global__ void merabetaKernel(int* a, int* b, int* c,int n ) {
	int i = threadIdx.x;
	if(i<n)
	c[i] = c[i] + a[i] + b[i];

}


void MatAdd(int * a,int *b,int *c,int size) {
	int *a, *b, *c;
	int size = 5;
	int id =0 ;
	hipError_t status;
	status= hipSetDevice(id);
	if (status != hipSuccess) {
		printf("ERROR no GPu");
		goto error;
	}
	
	hipMallocManaged(&a, 5);
	hipMallocManaged(&b, 5);
	hipMallocManaged(&c, 5);

	std::fill_n(a, 5, 6);
	std::fill_n(b, 5, 45);
	std::fill_n(c, 5, 20);

	b[2] = 78;
	
	hipMemPrefetchAsync(a, size, id);
	hipMemPrefetchAsync(b, size, id);
	hipMemPrefetchAsync(c, size, id);
	merabetaKernel <<< 1, size >>> (a, b, c,size);
	hipDeviceSynchronize();
	hipMemPrefetchAsync(c, size, hipCpuDeviceId);

	for (int i = 0; i < size; i++)
		printf("%d \n",c[i]);




error:
	hipFree(a);
	hipFree(b);
	hipFree(c);




}